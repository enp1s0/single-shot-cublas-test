#include <iostream>
#include <hipblas.h>
#include <chrono>
#include <type_traits>
#include <string>

#include <mateval/cuda/comparison.hpp>
#include <mateval/cuda/utils.hpp>

#include <cutf/hiprand.hpp>
#include <curand_fp16/curand_fp16.hpp>

constexpr unsigned num_test = 128;

template <class T>
inline hipDataType get_cuda_data_type();
template <>
inline hipDataType get_cuda_data_type<float>() {return HIP_R_32F;};
template <>
inline hipDataType get_cuda_data_type<hipComplex>() {return HIP_C_32F;};
template <>
inline hipDataType get_cuda_data_type<double>() {return HIP_R_64F;};
template <>
inline hipDataType get_cuda_data_type<hipDoubleComplex>() {return HIP_C_32F;};
template <>
inline hipDataType get_cuda_data_type<half>() {return HIP_R_16F;};

template <class T>
inline T one() {return 1;}
template <>
inline hipComplex one<hipComplex>() {return make_hipComplex(1, 0);}
template <>
inline hipDoubleComplex one<hipDoubleComplex>() {return make_hipDoubleComplex(1, 0);}

inline std::string get_op_str(const hipblasOperation_t op) {
  switch(op) {
    case HIPBLAS_OP_N:
      return "N";
    case HIPBLAS_OP_T:
      return "T";
    case HIPBLAS_OP_C:
      return "C";
    default:
      return "Unknown";
  }
}

struct run_gemm_base {
  virtual void operator()(
      hipblasOperation_t op_a,
      hipblasOperation_t op_b,
      const std::size_t m,
      const std::size_t n,
      const std::size_t k,
      const std::size_t lda,
      const std::size_t ldb,
      const std::size_t ldc
      ) = 0;
};

template <class T>
struct real_type {
  using type = T;
};

template <>
struct real_type<hipDoubleComplex> {
  using type = double;
};

template <>
struct real_type<hipComplex> {
  using type = float;
};

template <class T>
using real_type_v = typename real_type<T>::type;

template <class T>
struct run_gemm : run_gemm_base {
  void operator() (
      hipblasOperation_t op_a,
      hipblasOperation_t op_b,
      const std::size_t m,
      const std::size_t n,
      const std::size_t k,
      const std::size_t lda,
      const std::size_t ldb,
      const std::size_t ldc
      ) override {
    const auto mat_a_size = lda * (op_a == HIPBLAS_OP_N ? k : m);
    const auto mat_b_size = ldb * (op_b == HIPBLAS_OP_N ? n : k);
    const auto mat_c_size = ldc * n;

    T *mat_a, *mat_b, *mat_c, *mat_d;
    hipMalloc(&mat_a, mat_a_size * sizeof(T));
    hipMalloc(&mat_b, mat_b_size * sizeof(T));
    hipMalloc(&mat_c, mat_c_size * sizeof(T));
    hipMalloc(&mat_d, mat_c_size * sizeof(T));

    if constexpr (!std::is_same_v<T, half>) {
      auto cgen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_XORWOW);
      cutf::hiprand::generate_normal(*cgen.get(), reinterpret_cast<real_type_v<T>*>(mat_a), mat_a_size, 0, 1);
      cutf::hiprand::generate_normal(*cgen.get(), reinterpret_cast<real_type_v<T>*>(mat_b), mat_b_size, 0, 1);
      cutf::hiprand::generate_normal(*cgen.get(), reinterpret_cast<real_type_v<T>*>(mat_c), mat_c_size, 0, 1);
    } else {
      mtk::curand_fp16::generator_t cugen;
      mtk::curand_fp16::create(cugen, HIPRAND_RNG_PSEUDO_XORWOW);

      mtk::curand_fp16::normal(cugen, reinterpret_cast<real_type_v<T>*>(mat_a), mat_a_size, 0, 1);
      mtk::curand_fp16::normal(cugen, reinterpret_cast<real_type_v<T>*>(mat_b), mat_b_size, 0, 1);
      mtk::curand_fp16::normal(cugen, reinterpret_cast<real_type_v<T>*>(mat_c), mat_c_size, 0, 1);
    }

    hipMemset(mat_a, 0x0, mat_a_size * sizeof(T));
    hipMemset(mat_b, 0x0, mat_b_size * sizeof(T));
    hipMemset(mat_c, 0x0, mat_c_size * sizeof(T));
    hipMemcpy(mat_d, mat_c, mat_c_size * sizeof(T), hipMemcpyDefault);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    hipDeviceSynchronize();
    const auto start_clock = std::chrono::system_clock::now();

    const T alpha = one<T>(), beta = one<T>();

    hipblasGemmEx(
        cublas_handle,
        op_a,
        op_b,
        m, n, k,
        &alpha,
        mat_a, get_cuda_data_type<T>(), lda,
        mat_b, get_cuda_data_type<T>(), ldb,
        &beta,
        mat_d, get_cuda_data_type<T>(), ldc,
        get_cuda_data_type<T>(),
        HIPBLAS_GEMM_DEFAULT
        );
    const auto error = mtk::mateval::cuda::get_error_GEMM(
        mtk::mateval::relative_residual,
        m, n, k,
        mtk::mateval::utils::get_mateval_layout(op_a),
        mtk::mateval::utils::get_mateval_layout(op_b),
        mtk::mateval::col_major,
        mtk::mateval::col_major,
        alpha,
        mat_a, lda,
        mat_b, ldb,
        beta,
        mat_c, ldc,
        mat_d, ldc
        );

    for (unsigned t = 0; t < num_test; t++) {
      hipblasGemmEx(
          cublas_handle,
          op_a,
          op_b,
          m, n, k,
          &alpha,
          mat_a, get_cuda_data_type<T>(), lda,
          mat_b, get_cuda_data_type<T>(), ldb,
          &beta,
          mat_c, get_cuda_data_type<T>(), ldc,
          get_cuda_data_type<T>(),
          HIPBLAS_GEMM_DEFAULT
          );
    }

    hipDeviceSynchronize();
    const auto end_clock = std::chrono::system_clock::now();

    const auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9;
    auto complexity = 2lu * m * n * k;

    if (std::is_same_v<hipComplex, T> || std::is_same_v<hipDoubleComplex, T>) {
      complexity *= 4;
    }

    std::printf("op_A=%s, op_B=%s, shape=(%lu, %lu, %lu), ld=(%lu, %lu, %lu), throughput=%e TFlop/s, relative_error=%e\n",
                get_op_str(op_a).c_str(),
                get_op_str(op_b).c_str(),
                m, n, k,
                lda, ldb, ldc,
                complexity * num_test / elapsed_time * 1e-12,
                error.at(mtk::mateval::relative_residual)
               );

    hipblasDestroy(cublas_handle);

    hipFree(mat_a);
    hipFree(mat_b);
    hipFree(mat_c);
    hipFree(mat_d);
  }
};

inline hipblasOperation_t get_op(const std::string s) {
  if (s == "N" || s == "n") {
    return HIPBLAS_OP_N;
  } else if (s == "T" || s == "t") {
    return HIPBLAS_OP_T;
  } else if (s == "C" || s == "c") {
    return HIPBLAS_OP_C;
  } else {
    throw std::runtime_error("Unknown op: " + s);
  }
}

// ./cublas.out sgemm m n k
int main(int argc, char** argv) {
  if (argc < 7) {
    std::fprintf(stderr,
                 "%s [gemm type] [op_A] [op_B] [m] [n] [k] (optional: [lda] [ldb] [ldc])\n"
                 " - [gemm type] : s | c | d | z | h\n"
                 " - [op_A/B] : N | T | C\n",
                 argv[0]
                );
    return 1;
  }

  const std::string gemm_mode_str = argv[1];

  const auto op_a = get_op(argv[2]);
  const auto op_b = get_op(argv[3]);
  const auto m = std::stoul(argv[4]);
  const auto n = std::stoul(argv[5]);
  const auto k = std::stoul(argv[6]);

  const auto lda = (argc >= 8) ? std::stoul(argv[7]) : (op_a == HIPBLAS_OP_N ? m : k);
  const auto ldb = (argc >= 9) ? std::stoul(argv[8]) : (op_b == HIPBLAS_OP_N ? k : n);
  const auto ldc = (argc >= 10) ? std::stoul(argv[9]) : m;

  run_gemm_base* gemm = nullptr;
  if (gemm_mode_str == "s") {
    gemm = new run_gemm<float>;
  } else if (gemm_mode_str == "c") {
    gemm = new run_gemm<hipComplex>;
  } else if (gemm_mode_str == "d") {
    gemm = new run_gemm<double>;
  } else if (gemm_mode_str == "z") {
    gemm = new run_gemm<hipDoubleComplex>;
  } else if (gemm_mode_str == "h") {
    gemm = new run_gemm<half>;
  } else {
    std::fprintf(stderr, "Unknown gemm type : %s\n", gemm_mode_str.c_str());
    return 1;
  }

  (*gemm)(
      op_a,
      op_b,
      m, n, k,
      lda,
      ldb,
      ldc
      );

  delete gemm;
}
